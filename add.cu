
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(float *x, float *y)
{
    y[blockIdx.x] = x[blockIdx.x] + y[blockIdx.x];
    int a = y[blockIdx.x];
    char b = 0;
}

int main(void)
{
  const int N = 1<<20;
  std::cout << N << std::endl;

  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  std::cout << N << std::endl;
  for (int i = 0; i < N; i++) {
    maxError = fmax(maxError, fabs(y[i]-3.0f));
    // std::cout << maxError << std::endl;
  }
  hipDeviceSynchronize();

  std::cout << "Max error: " << maxError << std::endl;

  char a;
  scanf("%d", &a);

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}